#include "hip/hip_runtime.h"
/*

Copyright (c) 2017, NVIDIA Corporation
All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
this list of conditions and the following disclaimer in the documentation
and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
OF THE POSSIBILITY OF SUCH DAMAGE.

*/

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#warning "Use of all warp threads is TOO iffy without CUDA support for compute_70 or above. Will run 1 thread per warp."
#endif

#include <sys/mman.h>
#include <cuda/launch>
#include <cuda/atomic>
#include <cuda/semaphore>
#include <cassert>
#include <mutex>
#include <thread>
#include <iostream>

#define __test_abi __host__ __device__

template<class T>
using atomic = cuda::experimental::atomic<T>;
using thread = std::thread;
using binary_semaphore = cuda::experimental::binary_semaphore;
using counting_semaphore = cuda::experimental::counting_semaphore;
using condition_variable_atomic = cuda::experimental::condition_variable_atomic;
namespace details = cuda::experimental::details;

#include "test.hpp"

using mutex = binary_semaphore_mutex;

template<class F>
__global__ void run_gpu_thread(uint32_t count, uint32_t count_per_block, F const* f) {

    unsigned int const myIdx = blockIdx.x * count_per_block + threadIdx.x;
    if (myIdx < count)
        (*f)(myIdx);
}

uint32_t cap = 0;
bool use_malloc_managed = true;
uint32_t max_block_count = 0;

void* allocate_raw_bytes(size_t s, bool force = false) { 
    void* ptr = nullptr;
#ifdef __HIPCC__
    if(use_malloc_managed || force) {
        auto const ret = cap < 6 ? hipHostAlloc(&ptr, s, 0) : hipMallocManaged(&ptr, s);
        assert(ret == hipSuccess);
        if(cap >= 6)
            hipMemAdvise(ptr, s, hipMemAdviseSetPreferredLocation, 0);
    }
    else
#endif
    {
        ptr = mmap(0, s, PROT_READ|PROT_WRITE, MAP_PRIVATE|MAP_ANONYMOUS, -1, 0);
        if(ptr == MAP_FAILED) {
            ptr = nullptr;
        }
    }
    assert(ptr != nullptr);
    return ptr;
}

void deallocate_raw_bytes(void* ptr, bool force = false) {
#ifdef __HIPCC__
    if(use_malloc_managed || force)
        hipFree(ptr);
    else
#endif
    {
        // Leak. Oops.
    }
}

template<class F>
F* start_gpu_threads(uint32_t count, F f) {

    if(!count)
        return nullptr;

    uint32_t const blocks = (std::min)(count, max_block_count);
    uint32_t const threads_per_block = (count / blocks) + (count % blocks ? 1 : 0);

    auto const fptr = new (allocate_raw_bytes(sizeof(F))) F(f);
    assert(uintptr_t(fptr) % alignof(F) == 0);
    run_gpu_thread<F><<<blocks, threads_per_block>>>(count, threads_per_block, fptr);

    return fptr;
}

template<class F>
void stop_gpu_threads(F* fptr) {
    if(nullptr == fptr)
        return;
    auto const ret = hipDeviceSynchronize();
    assert(ret == hipSuccess);
    fptr->~F();
    deallocate_raw_bytes(fptr);
}

uint32_t dev = 0;

unsigned int max_gpu_threads() { 

    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    max_block_count = deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor / 1024;
    cap = deviceProp.major;
    if (cap < 7)
        return max_block_count * 32;
    else
        return max_block_count * 32 * 32;
}

#include "driver.cpp"

#ifndef TEST_NO_MAIN

int main(int argc, char const* argv[]) {

    auto f = [](uint32_t&,double&) { };

    return driver_main(argc, argv, f);
}

#endif